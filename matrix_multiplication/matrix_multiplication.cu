#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#define BLOCK_SIZE 16

//this function is for filling the matrices with cos and sin values randomly
//I transform the matrices to square matrix in order to perform better multiplication
__host__ int fill(float **Lmatrix, float **Rmatrix, int LdimX, int LdimY, int RdimX, int RdimY) {

    int sqr_dim_X, sqr_dim_Y, size;

    sqr_dim_X = RdimX;
    if (LdimX > RdimX) {
        sqr_dim_X = LdimX;
    }

    sqr_dim_Y = RdimY;
    if (LdimY > RdimY) {
        sqr_dim_Y = LdimY;
    }

    size = sqr_dim_Y;
    if (sqr_dim_X > sqr_dim_Y) {
        size = sqr_dim_X;
    }

    int temp = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
    size = temp * BLOCK_SIZE;

    size_t pt_size = size * size * sizeof(float);

    *Lmatrix = (float *) malloc(pt_size);
    *Rmatrix = (float *) malloc(pt_size);

    memset(*Lmatrix, 0, pt_size);
    memset(*Rmatrix, 0, pt_size);

    for (int i = 0; i < LdimX; i++) {
        for (int j = 0; j < LdimY; j++) {
            int dummy = size * i + j;
            (*Lmatrix)[dummy] = sinf(dummy);
        }
    }
    for (int i = 0; i < RdimX; i++) {
        for (int j = 0; j < RdimY; j++) {
            int dummy = size * i + j;
            (*Rmatrix)[dummy] = cosf(dummy);
        }
    }
    return size;
}

// Kernel that executes on the CUDA device
/* left: left operand
 * right: right operand
 * res : result array
 * dim: M dimension of MxM matrix
 * Blok_size: defines block size
 *
 * this function divides the matrices to tiles and load those tiles to shared memory
 * After loading to shared memory it function multiplies with the corresponding tile of other matrix
 * After finishing multiplication of 1 row and 1 column by collecting results of different tiles
 * it stores the result in global memory
 * Function has coalesced access to the global memory and prevent bank conflict
 */
__global__ void multiply(float *left, float *right, float *res, int dim) {

    int i,j;
    float temp = 0;

    __shared__ float Left_shared_t [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

    // Row i of matrix left
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

        // Column j of matrix left
        j = tileNUM * BLOCK_SIZE + threadIdx.x;
        i = tileNUM * BLOCK_SIZE + threadIdx.y;
        // Load left[i][j] to shared mem

        Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];// Coalesced access
        // Load right[i][j] to shared mem

        Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col]; // Coalesced access
        // Synchronize before computation
        __syncthreads();

        // Accumulate one tile of res from tiles of left and right in shared mem
        for (int k = 0; k < BLOCK_SIZE; k++) {

            temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x]; //no shared memory bank conflict
        }
        // Synchronize
        __syncthreads();
    }
    // Store accumulated value to res
    res[row * dim + col] = temp;
}

// main routine that executes on the host
int main(void)
{
    //size of the vectors to be processed  and matrix dimensions
    int Left_matrix_x = 8192;
    int Left_matrix_y = 8192;
    int Right_matrix_x = 8192;
    int Right_matrix_y = 8192;

    float *Left_Vector_h, *Right_Vector_h, *Left_Vector_d, *Right_Vector_d, *Res_d;  // Pointer to host & device arrays

    int dim = fill(&Left_Vector_h, &Right_Vector_h, Left_matrix_x, Left_matrix_y, Right_matrix_x, Right_matrix_y); //fills the matrices with random values


    size_t vector_size;
    vector_size = dim*dim * sizeof(float);

    hipMalloc((void **) &Left_Vector_d, vector_size);     // Allocate array on device for LHS operand
    hipMalloc((void **) &Right_Vector_d, vector_size);   // Allocate array on device for RHS operand but this is vector 1xN
    hipMalloc((void **) &Res_d, vector_size);     // Allocate array on device for result

    hipMemcpy(Left_Vector_d, Left_Vector_h, vector_size, hipMemcpyHostToDevice);      // copy values to device
    hipMemcpy(Right_Vector_d, Right_Vector_h, vector_size, hipMemcpyHostToDevice);   // copy values to device

    //Block dimension is directly from block_size
    dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
    //Grid dimension is found by dividing matrix dimension to block_size
    dim3 Grid_dim(dim / BLOCK_SIZE, dim / BLOCK_SIZE);

    //commented out the functions which helps to calculate time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //kernel call
    multiply << < Grid_dim, Block_dim >> > (Left_Vector_d, Right_Vector_d, Res_d, dim);

    //commented out the functions which helps to calculate time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //commented out the functions which helps to calculate time
    printf("GPU time= %f ms\n", et);
    printf("\n");

    // Cleanup
    free(Left_Vector_h);
    free(Right_Vector_h);
    hipFree(Left_Vector_d);
    hipFree(Right_Vector_d);
    hipFree(Res_d);
}